
#include <hip/hip_runtime.h>

#include <stdio.h>
/*
enum Type {
	NOT_NODE = 0,
	AND_NODE,
	IMPLIES_NODE,
	UNIVERSAL_NODE,
	EXISTENTIAL_NODE,
	BFUNC_NODE,
	EMPTY_NODE,
	SAME,
	SZ_SPD_CLOSE,
	SZ_LOC_CLOSE,
	SZ_LOC_DIST,
	SZ_LOC_DIST_NEQ ,
	SZ_LOC_RANGE,
	OR_NODE
};*/

#define NOT_NODE 0
#define AND_NODE 1
#define IMPLIES_NODE 2
#define UNIVERSAL_NODE 3
#define EXISTENTIAL_NODE 4
#define BFUNC_NODE 5
#define EMPTY_NODE 6
#define STABLE_DR_V 7
#define STABLE_SPEED_VARIANCE 8
#define OR_NODE 9

#define MAX_PARAM_NUM 2
#define MAX_CCT_SIZE 3000000
#define MAX_LINK_SIZE 5000
#define DEBUG

struct Context{
	int id;
	double DR_V;
	double DR_I;
	double speed;
	double timestamps;
};

struct Node {
	Node *next;
	Node *tail;
	int params[MAX_PARAM_NUM];
};


__device__ bool truth_values[MAX_CCT_SIZE];
__device__ Node links[MAX_CCT_SIZE];



extern "C"
__device__ bool stable_DR_V(Context c){
	return c.DR_V >= 1450.0 && c.DR_V <= 1800.0;
}

extern "C"
__device__ bool stable_speed_variance(Context c1, Context c2){
    double speed_delta = (c1.speed - c2.speed)*1000.0/3600.0;
	double timestamps_delta = (c1.timestamps - c2.timestamps)/1000.0;
	if (timestamps_delta==0)
	    return true;
	double speed_variance = speed_delta/timestamps_delta;
	if(speed_variance<0)
	    speed_variance = speed_variance*-1;
	return speed_variance >= 0.0 && speed_variance <= 0.95;
}

extern "C"
__device__ void init_node(Node *n){
	n->next = NULL;
	n->tail = n;
	for (int i = 0; i < MAX_PARAM_NUM; i++) {
		n->params[i] = -1;
	}
}

extern "C"
__device__ bool is_null_node(Node *n){
	bool res = true;
	for (int i = 0; i < MAX_PARAM_NUM; i++) {
		res = res && (n->params[i] == -1);
	}
	return res;
}


extern "C"
__device__ void linkHelper(Node *link1, Node *link2) {
	//inital and assumpt that link1 != null, links != null
	if (is_null_node(link1)) {
		for (int i = 0; i < MAX_PARAM_NUM; i++) {
			link1->params[i] = link2->params[i];
		}
		link1->next = NULL;
		link1->tail = link1;


		if(link2->next != NULL) {
			link2->next->tail = link2->tail;
		}
		link2 = link2->next;
	}

	if (link2 == NULL) {
		return;
	}

	link1->tail->next = link2;
	link1->tail = link2->tail;
}

extern "C"
__device__ int calc_offset(	int node, int tid, Context *params,
							int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx,
							int *pattern_begin, int *pattern_length, int *pattern,
							double *DR_V, double *DR_I, double *speed, double *timestamps, // contexts
							int *branch_size) {

	int offset = branch_size[node];
	int current_node = node;
	int index = 0, tmp = tid;
	while (parent[current_node] != -1) {
		int type = node_type[parent[current_node]];
		if (type == EXISTENTIAL_NODE || type == UNIVERSAL_NODE) {
			int len = pattern_length[pattern_idx[parent[current_node]]];
			int branch_idx = tmp % len;
			tmp /= len;

			params[index].id = pattern[pattern_begin[pattern_idx[parent[current_node]]] + branch_idx];//(pattern + pattern_idx[parent[current_node]] * MAX_PATTERN_SIZE)[(branch_idx + pattern_begin[pattern_idx[parent[current_node]]]) % MAX_PATTERN_SIZE];
			params[index].DR_V = DR_V[params[index].id];
			params[index].DR_I = DR_I[params[index].id];
			params[index].speed = speed[params[index].id];
			params[index].timestamps = timestamps[params[index].id];

			offset += branch_idx * branch_size[current_node] ;
//			printf("branch_idx = %d, branch_size = %d\n", branch_idx, branch_size[current_node]);
			index++;
		}
		else if (type == AND_NODE || type == IMPLIES_NODE || type == OR_NODE) {
			if (right_child[parent[current_node]] == current_node) {
				offset += branch_size[left_child[parent[current_node]]];
			}
		}
		else {
		    offset += 0;
		}
		current_node = parent[current_node];
	}
	return offset - 1;
}

extern "C"
__global__ void evaluation(int *parent, int *left_child, int *right_child, int *node_type, int *pattern_idx, //constraint rule
                          	 int *branch_size, int cunit_begin, int cunit_end,//cunit_end is the root of cunit
                          	 int *pattern_begin, int *pattern_length, int *pattern, //patterns
							 double *DR_V, double *DR_I, double *speed, double *timestamps, // contexts
                          	 short *truth_value_result,
                          	 int *link_result, int *link_num, int *cur_link_size,
                          	 int last_cunit_root,
                          	 int ccopy_num) {


	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < ccopy_num) {

		Context params[MAX_PARAM_NUM];
		for (int i = 0; i < MAX_PARAM_NUM; i++) {
            params[i].id = -1;
         }
		int ccopy_root_offset = calc_offset(cunit_end, tid, params,
											parent, left_child, right_child, node_type, pattern_idx,
											pattern_begin, pattern_length, pattern,
											DR_V, DR_I, speed, timestamps,
											branch_size);

//#ifdef DEBUG
//		printf("root = %d, ccopynum = %d, offset = %d\n",cunit_end, ccopy_num, ccopy_root_offset);
//#endif
		for (int node = cunit_begin; node <= cunit_end; node++) {
			int offset = ccopy_root_offset - (cunit_end - node);
			int type = node_type[node];
			bool value;

			Node* cur_links = &links[offset];
			init_node(cur_links);

			switch(type) {
				case UNIVERSAL_NODE: {
					int step = branch_size[left_child[node]];
					value = true;
					bool first = true;
					for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
						value = value && truth_values[offset - (i * step + 1)];
						if(!truth_values[offset - (i * step + 1)]) {
							if(first) {
								init_node(cur_links);
								first = false;
							}
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
						else if(value) {
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
					}

					break;
				}

				case EXISTENTIAL_NODE: {
					int step = branch_size[left_child[node]];
					value = false;
					bool first = true;
					for (int i = 0; i < pattern_length[pattern_idx[node]]; i++) {
						value = value || truth_values[offset - (i * step + 1)];
						if(truth_values[offset - (i * step + 1)]) {
							if(first) {
								init_node(cur_links);
								first = false;
							}
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
						else if(!value) {
							linkHelper(cur_links, &(links[offset - (i * step + 1)]));
						}
					}
					break;
				}

				case AND_NODE: {
					//right && left
					value = truth_values[offset - 1] && truth_values[offset - (branch_size[right_child[node]] + 1)];

					if (truth_values[offset - 1] == value) {
						linkHelper(cur_links, &(links[offset - 1]));
					}

					if (truth_values[offset - (branch_size[right_child[node]] + 1)] == value) {
						linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}

					break;
				}
				case OR_NODE: {
					//right || left
					value = truth_values[offset - 1] || truth_values[offset - (branch_size[right_child[node]] + 1)];

					if (truth_values[offset - 1] == value) {
						linkHelper(cur_links, &(links[offset - 1]));
					}

					if (truth_values[offset - (branch_size[right_child[node]] + 1)] == value) {
						linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}

					break;
				}

				case IMPLIES_NODE: {
					//!left || right
					value = !truth_values[offset - (branch_size[right_child[node]] + 1)] || truth_values[offset - 1];

					if(value) {
	                   linkHelper(cur_links, &(links[offset - 1]));
	                   linkHelper(cur_links, &(links[offset - (branch_size[right_child[node]] + 1)]));
					}
					else {
					   linkHelper(cur_links, &(links[offset - 1]));
					}

					break;
				}

				case NOT_NODE: {
					value = !truth_values[offset - 1];
					linkHelper(cur_links, &(links[offset - 1]));
					break;
				}

				default : { //BFUNC
					switch(type) {
						case STABLE_DR_V: {
							value = stable_DR_V(params[0]);
							break;
						}

						case STABLE_SPEED_VARIANCE: {
							value = stable_speed_variance(params[0], params[1]);
							break;
						}
					}

			
					for (int i = 0; i < MAX_PARAM_NUM; i++) {
						cur_links->params[i] = params[i].id;
					}
					break;
				}

				
			}

			truth_values[offset] = value;
		}

		if (last_cunit_root == cunit_end ) {
		    *truth_value_result = truth_values[ccopy_root_offset];
		    if(!truth_values[ccopy_root_offset]) {
            
         		int len = 0;
                for(Node *head = &links[ccopy_root_offset]; head != NULL; head = head ->next) {
                
                	if(len < MAX_LINK_SIZE) {
	                	for(int j = 0; j < MAX_PARAM_NUM; j++) {
	                         link_result[MAX_PARAM_NUM * len + j] = head->params[j];
	                    }
                	}

                    len++;
                }
                
                *cur_link_size = len;
                *link_num = len > MAX_LINK_SIZE ? MAX_LINK_SIZE : len;
         	}
        }
	}
 }
